#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include "config.h"
#include "bench.h"
#include "gpu_bench.h"
#include "hip/hip_runtime_api.h"


unsigned char *gpu_array_make_uma(unsigned int bytes)
{
	unsigned char *array;

	checkCudaErrors(hipMallocManaged(&array, bytes));

	return array;
}

float test_host_to_device_uma(unsigned char *array, unsigned int bytes, int loops)
{
	float elapse = 0;
	float bandwidth = 0;
	hipEvent_t start, stop;
	int l = loops;        
	unsigned char value = rand() % 256;
	
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	gpu_test_status = HOST_TO_DEVICE;
	pthread_barrier_wait(&gpu_barrier);

	checkCudaErrors(hipEventRecord(start, 0));
	while(l--) {
		for(unsigned int i = 0; i < bytes / sizeof(unsigned char); i++) {
			array[i] = value;
		}
	}

	checkCudaErrors(hipEventRecord(stop, 0));

	checkCudaErrors(hipEventSynchronize(start));
	checkCudaErrors(hipEventSynchronize(stop));

	gpu_test_status = HOST_TO_DEVICE_COMPLETE;
	pthread_barrier_wait(&gpu_barrier);
	
 	checkCudaErrors(hipEventElapsedTime(&elapse, start, stop));

	printf("elapse is %fms, bytes: %u\n", elapse, bytes * loops);
	bandwidth =  ((long)bytes * loops / MB) / (elapse / 1000);
	
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipEventDestroy(start));

	return bandwidth;
}

void gpu_array_destroy(unsigned char *array)
{
	hipFree(array);
}

__global__ void gpu_array_read(unsigned char *array, unsigned int size)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned char value;
	if(x < size) {
		value = array[x];
	}
}

float test_device_access(unsigned char *array, unsigned int size)
{
	int threads = 1024;
	int blocks = (size - 1) / threads + 1;
	float elapse = 0;
	float bandwidth = 0;
	hipEvent_t start, stop;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	gpu_test_status = DEVICE;
	pthread_barrier_wait(&gpu_barrier);

	checkCudaErrors(hipEventRecord(start, 0));
	gpu_array_read<<<blocks, threads>>>(array, size);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipEventRecord(stop, 0));

	checkCudaErrors(hipEventSynchronize(start));
	checkCudaErrors(hipEventSynchronize(stop));
	gpu_test_status = DEVICE_COMPLETE;
	pthread_barrier_wait(&gpu_barrier);
	
	checkCudaErrors(hipEventElapsedTime(&elapse, start, stop));

	printf("elapse is %fms\n", elapse);
	bandwidth =  ((long)size / MB) / (elapse / 1000);
	
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipEventDestroy(start));

	return bandwidth;
}

extern "C"
void gpu_bench(struct config *con)
{
	unsigned int bytes = con->gpu_array_size * sizeof(unsigned char); 
	float bandwidth;
	gpu_test_status = INIT;
	pthread_barrier_wait(&gpu_barrier);
	unsigned char *array = gpu_array_make_uma(con->gpu_array_size * sizeof(unsigned char));
	bandwidth = test_host_to_device_uma(array, bytes, con->loops);
	printf("Host to device: %.3f MiB/s\n", bandwidth);

	bandwidth = test_device_access(array, bytes);
	printf("device access memory: %.3f MiB/s\n", bandwidth);

	gpu_array_destroy(array);
	gpu_test_status = COMPLETE;
	pthread_barrier_wait(&gpu_barrier);
}



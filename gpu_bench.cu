#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include "bench.h"
#include "gpu_bench.h"
#include "hip/hip_runtime_api.h"


unsigned char *gpu_array_make_uma(unsigned int bytes)
{
	unsigned char *array;

	checkCudaErrors(hipMallocManaged(&array, bytes));

	return array;
}

float test_host_to_device_uma(unsigned char *array, unsigned int bytes, int loops)
{
	float elapse = 0;
	float bandwidth = 0;
	hipEvent_t start, stop;
	int l = loops;        
	unsigned char value = rand() % 256;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));
	while(l--) {
		for(unsigned int i = 0; i < bytes / sizeof(unsigned char); i++) {
			array[i] = value;
		}
	}
	checkCudaErrors(hipEventRecord(stop, 0));

	checkCudaErrors(hipEventSynchronize(start));
	checkCudaErrors(hipEventSynchronize(stop));
	gpu_done = 1;
 	checkCudaErrors(hipEventElapsedTime(&elapse, start, stop));
	printf("bytes: %d, elapse: %f, loops: %d\n", bytes, elapse, loops);	
	bandwidth =  ((long)bytes * loops / MB) / (elapse / 1000);
	
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipEventDestroy(start));

	return bandwidth;
}

void gpu_array_destroy(unsigned char *array)
{
	hipFree(array);
}

extern "C"
void gpu_bench(struct config *con)
{
	unsigned int bytes = con->size * sizeof(long);
	float bandwidth;
	unsigned char *array = gpu_array_make_uma(con->size * sizeof(long));
	bandwidth = test_host_to_device_uma(array, bytes, con->loops);
	printf("Host to device: %.3f MiB/s\n", bandwidth);
	gpu_array_destroy(array);
}

void test()
{

}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "bench.h"
#include "gpu_bench.h"
#include "hip/hip_runtime_api.h"


unsigned char *gpu_array_make_uma(unsigned int bytes)
{
	unsigned char *array;

	checkCudaErrors(hipMallocManaged(&array, bytes));

	return array;
}

double test_host_to_device_uma(unsigned char *array, unsigned int bytes, int loops)
{
	float elapse = 0;
	float bandwidth = 0;
	hipEvent_t start, stop;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));
	while(loops--) {
		for(unsigned int i = 0; i < bytes / sizeof(unsigned char); i++) {
			array[i] = rand() % 256;
		}
	}
	checkCudaErrors(hipEventRecord(stop, 0));

	checkCudaErrors(hipEventSynchronize(start));
	checkCudaErrors(hipEventSynchronize(stop));
 	checkCudaErrors(hipEventElapsedTime(&elapse, start, stop));
	
	bandwidth =  ((float)(1<<10) * bytes * loops) /
                     (elapse * (float)(1 << 20));
	
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipEventDestroy(start));

	return bandwidth;
}

void gpu_array_destroy(unsigned char *array)
{
	hipFree(array);
}

void gpu_bench(struct config *con)
{
	unsigned int bytes = con->size * sizeof(long);
	double bandwidth;
	unsigned char *array = gpu_array_make_uma(con->size * sizeof(long));
	bandwidth = test_host_to_device_uma(array, bytes, con->loops);
	printf("Host to device: %.3f MiB/s\n", bandwidth);
	gpu_array_destroy(array);
}

